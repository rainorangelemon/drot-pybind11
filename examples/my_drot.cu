#include "hip/hip_runtime.h"
#include <vector>
#include <string>
#include <iostream>
#include "drot.hpp"
#include "reader.hpp"

int main(int argc, char *argv[]) {
    if (argc != 6) {
        std::cerr << "Usage: " << argv[0] << " input_file output_file nrows ncols maxiters\n";
        return 1;
    }

    const std::string input_file = argv[1];
    const std::string output_file = argv[2];
    const int nrows = std::stoi(argv[3]);
    const int ncols = std::stoi(argv[4]);
    const int maxiters = std::stoi(argv[5]);

    auto C = utility::load<float>(input_file, nrows, ncols);
    std::vector<float> p(nrows, 1/(float) nrows);
    std::vector<float> q(ncols, 1/(float) ncols);

    const float stepsize = 2/(float) (nrows + ncols);
    const float eps = 0.0f;

    const bool verbose = true;
    const bool log = true;

    drot(&C[0], &p[0], &q[0], nrows, ncols, stepsize, maxiters, eps, verbose, log, output_file);

    return 0;
}