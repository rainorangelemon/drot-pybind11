#include "hip/hip_runtime.h"
#include <math.h>
#include <vector>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <tuple>

#define BLOCK_SIZE 128
#define WORK_SIZE 1

namespace py = pybind11;

template<typename T>
__inline__ __device__ T warp_reduce_sum(T val) {
  for (int w = 16; w > 0; w /= 2)
      val += __shfl_down_sync(0xffffffff, val, w);
  return val;
}

template<typename T>
__inline__ __device__ T block_reduce_sum(T val) {
    static __shared__ T shared[32];

    val = warp_reduce_sum(val);
    if (threadIdx.y % 32==0)
        shared[threadIdx.y / 32] = val;
    __syncthreads();

    val = (threadIdx.y < blockDim.y / 32) ? shared[threadIdx.y % 32] : 0;
    if (threadIdx.y / 32==0)
        val = warp_reduce_sum(val);

    return val;
}

template<typename T>
__global__ void update_x_even(T *X,
        T *a2,
        T *b2,
        T *gamma2,
        T *loginfo,
        const T * __restrict__ phi1,
        const T * __restrict__ phi2,
        const T * __restrict__ C,
        const T stepsize,
        const int nrows,
        const int ncols) {
    const int tidy = threadIdx.y + blockIdx.y * blockDim.y;
    int offset;

    __shared__ T phi2_shared[BLOCK_SIZE];
    T xval=0;
    T colsum=0;
    const T phi1_val=phi1[tidy];

    for (int idx=0; idx<WORK_SIZE; idx++) {
        offset = (blockIdx.x * WORK_SIZE + idx) * BLOCK_SIZE; // Row offset
        if (threadIdx.y + offset < ncols)
            phi2_shared[threadIdx.y] = phi2[threadIdx.y + offset];
        __syncthreads();

        if (tidy < nrows) {
            if (BLOCK_SIZE + offset <= ncols) {
                #pragma unroll
                for (int e=0; e<BLOCK_SIZE; e++) {
                    xval = X[tidy + (e + offset) * nrows];
                    xval = max(xval + phi1_val + phi2_shared[e], T(0));
                    X[tidy + (e + offset) * nrows] = xval;

                    colsum += xval;
                    xval = warp_reduce_sum(xval);
                    if (threadIdx.y % 32==0)
                        atomicAdd(&b2[e + offset], xval);
                }
            } else { // Edge block
                #pragma unroll
                for (int e=ncols-1; e>=offset; e--) {
                    xval = X[tidy + e * nrows];
                    xval = max(xval + phi1_val + phi2_shared[e-offset], T(0));
                    X[tidy + e * nrows] = xval;

                    colsum += xval;
                    xval = warp_reduce_sum(xval);
                    if (threadIdx.y % 32==0)
                        atomicAdd(&b2[e], xval);
                }
            }
        }
    }
    if (tidy < nrows) {
        atomicAdd(&a2[tidy], colsum);
        colsum = warp_reduce_sum(colsum);
        if (threadIdx.y % 32==0)
            atomicAdd(gamma2, colsum);
    }
    if (tidy>=blockDim.y*gridDim.y-2 && blockIdx.x==0)
        loginfo[blockDim.y*gridDim.y - tidy - 1] = T(0);
}

template<typename T>
__global__ void update_x_odd(T *X,
        T *a2,
        T *b2,
        T *gamma2,
        T *loginfo,
        const T * __restrict__ phi1,
        const T * __restrict__ phi2,
        const T * __restrict__ C,
        const T stepsize,
        const int nrows,
        const int ncols) {
    const int tidy = threadIdx.y + blockIdx.y * blockDim.y;
    int offset;

    __shared__ T phi2_shared[BLOCK_SIZE];
    T xval=0;
    T cval=0;
    T colsum=0;
    const T phi1_val=phi1[tidy];

    for (int idx=0; idx<WORK_SIZE; idx++) {
        offset = (blockIdx.x * WORK_SIZE + idx) * BLOCK_SIZE; // Row offset
        if (threadIdx.y + offset < ncols)
            phi2_shared[threadIdx.y] = phi2[threadIdx.y + offset];
        __syncthreads();

        if (tidy < nrows) {
            if (BLOCK_SIZE + offset <= ncols) {
                #pragma unroll
                for (int e=0; e<BLOCK_SIZE; e++) {
                    xval = X[tidy + (e + offset) * nrows];
                    cval = stepsize * C[tidy + (e + offset) * nrows];
                    xval = max(xval + phi1_val + phi2_shared[e] - cval, T(0));
                    X[tidy + (e + offset) * nrows] = xval - cval;

                    colsum += xval;
                    xval = warp_reduce_sum(xval);
                    if (threadIdx.y % 32==0)
                        atomicAdd(&b2[e + offset], xval);
                }
            } else { // Edge blocks
                #pragma unroll
                for (int e=ncols-1; e>=offset; e--) {
                    xval = X[tidy + e * nrows];
                    cval = stepsize * C[tidy + e * nrows];
                    xval = max(xval + phi1_val + phi2_shared[e-offset]
                            - cval, T(0));
                    X[tidy + e * nrows] = xval - cval;

                    colsum += xval;
                    xval = warp_reduce_sum(xval);
                    if (threadIdx.y % 32==0)
                        atomicAdd(&b2[e], xval);
                }
            }
        }
    }
    if (tidy < nrows) {
        atomicAdd(&a2[tidy], colsum);
        colsum = warp_reduce_sum(colsum);
        if (threadIdx.y % 32==0)
            atomicAdd(gamma2, colsum);
    }
    if (tidy>=blockDim.y*gridDim.y-2 && blockIdx.x==0)
        loginfo[blockDim.y*gridDim.y - tidy - 1] = T(0);
}

template<typename T>
__global__ void update_x_even_(T *X,
        T *a2,
        T *b2,
        T *gamma2,
        T *loginfo,
        const T * __restrict__ phi1,
        const T * __restrict__ phi2,
        const T * __restrict__ C,
        const T stepsize,
        const int nrows,
        const int ncols) {
    const int tidy = threadIdx.y + blockIdx.y * blockDim.y;
    int offset;

    __shared__ T phi2_shared[BLOCK_SIZE];
    T xval=0;
    T colsum=0;
    const T phi1_val=phi1[tidy];

    for (int idx=0; idx<WORK_SIZE; idx++) {
        offset = (blockIdx.x * WORK_SIZE + idx) * BLOCK_SIZE; // Column offset
        if (threadIdx.y + offset < ncols)
            phi2_shared[threadIdx.y] = phi2[threadIdx.y + offset];
        __syncthreads();

        if (tidy < nrows) {
            if (BLOCK_SIZE + offset <= ncols) {
                #pragma unroll
                for (int e=0; e<BLOCK_SIZE; e++) {
                    xval = X[tidy + (e + offset) * nrows];
                    xval = max(xval + phi1_val + phi2_shared[e], T(0));
                    X[tidy + (e + offset) * nrows] = xval;

                    colsum += xval;
                    xval = warp_reduce_sum(xval);
                    if (threadIdx.y % 32==0)
                        atomicAdd(&b2[e + offset], xval);
                }
            } else { // Edge blocks
                #pragma unroll
                for (int e=ncols-1; e>=offset; e--) {
                    xval = X[tidy + e * nrows];
                    xval = max(xval + phi1_val + phi2_shared[e-offset], T(0));
                    X[tidy + e * nrows] = xval;

                    colsum += xval;
                    xval = warp_reduce_sum(xval);
                    if (threadIdx.y % 32==0)
                        atomicAdd(&b2[e], xval);
                }
            }
        }
    }
    if (tidy < nrows) {
        atomicAdd(&a2[tidy], colsum);
        colsum = warp_reduce_sum(colsum);
        if (threadIdx.y % 32==0)
            atomicAdd(gamma2, colsum);
    }
    if (tidy>=blockDim.y*gridDim.y-5 && blockIdx.x==0)
        loginfo[blockDim.y*gridDim.y - tidy - 1] = T(0);
}

template<typename T>
__global__ void update_x_odd_(T *X,
        T *a2,
        T *b2,
        T *gamma2,
        T *loginfo,
        const T * __restrict__ phi1,
        const T * __restrict__ phi2,
        const T * __restrict__ C,
        const T stepsize,
        const int nrows,
        const int ncols) {
    const int tidy = threadIdx.y + blockIdx.y * blockDim.y;
    int offset;

    __shared__ T phi2_shared[BLOCK_SIZE];
    T xval=0;
    T cval=0;
    T fval=0;
    T colsum=0;
    const T phi1_val=phi1[tidy];

    for (int idx=0; idx<WORK_SIZE; idx++) {
        offset = (blockIdx.x * WORK_SIZE + idx) * BLOCK_SIZE; // Column offset
        if (threadIdx.y + offset < ncols)
            phi2_shared[threadIdx.y] = phi2[threadIdx.y + offset];
        __syncthreads();

        if (tidy < nrows) {
            if (BLOCK_SIZE + offset <= ncols) {
                #pragma unroll
                for (int e=0; e<BLOCK_SIZE; e++) {
                    xval = X[tidy + (e + offset) * nrows];
                    cval = C[tidy + (e + offset) * nrows];
                    xval = max(xval + phi1_val + phi2_shared[e]
                            - stepsize * cval, T(0));
                    X[tidy + (e + offset) * nrows] = xval - stepsize * cval;

                    colsum += xval;
                    fval += xval * cval;
                    xval = warp_reduce_sum(xval);
                    if (threadIdx.y % 32==0)
                        atomicAdd(&b2[e + offset], xval);
                }
            } else {
                #pragma unroll
                for (int e=ncols-1; e>=offset; e--) {
                    xval = X[tidy + e * nrows];
                    cval = C[tidy + e * nrows];
                    xval = max(xval + phi1_val + phi2_shared[e-offset]
                            - stepsize * cval, T(0));
                    X[tidy + e * nrows] = xval - stepsize * cval;

                    colsum += xval;
                    fval += xval * cval;
                    xval = warp_reduce_sum(xval);
                    if (threadIdx.y % 32==0)
                        atomicAdd(&b2[e], xval);
                }
            }
        }
    }
    if (tidy < nrows) {
        atomicAdd(&a2[tidy], colsum);
        colsum = warp_reduce_sum(colsum);
        fval = warp_reduce_sum(fval);
        if (threadIdx.y % 32==0) {
            atomicAdd(gamma2, colsum);
            atomicAdd(&loginfo[2], fval);
        }
    }
    if (tidy>=blockDim.y*gridDim.y-2 && blockIdx.x==0) {
        loginfo[blockDim.y*gridDim.y - tidy - 1] = T(0);
    }
}

template<typename T>
__global__ void update_auxs_even(T *a1,
        T *b1,
        T *a2,
        T *b2,
        T *gamma1,
        T *gamma2,
        T *loginfo,
        T *phi1,
        T *phi2,
        const T* __restrict__ p,
        const T* __restrict__ q,
        const int nrows,
        const int ncols) {
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    T val1=*gamma1; // this way gamma will not be modified when updating
    T val2=*gamma2; // gamma1 and gamma2 by tid=0 && blockIdx.y=0
    T gamma;

    if (tid < nrows && blockIdx.y==0) {
        gamma = (2*(val2 - T(1)) - val1) / (T) (nrows + ncols);
        val1 = a1[tid];
        val2 = a2[tid] - p[tid];
        phi1[tid] = (val1 - 2*val2 + gamma) / (T) ncols;
        a1[tid] = val1 - val2;

        a2[tid] = T(0); // Reset for the next reduction

        val2 *= val2;
        val2 = warp_reduce_sum(val2);
        if (threadIdx.x % 32==0)
            atomicAdd(&loginfo[0], val2);
    }
    if (tid < ncols && blockIdx.y==1) {
        gamma = (2*(val2 - T(1)) - val1) / (T) (nrows + ncols);
        val1 = b1[tid];
        val2 = b2[tid] - q[tid];
        phi2[tid] = (val1 - 2*val2 + gamma) / (T) nrows;
        b1[tid] = val1 - val2;

        b2[tid] = T(0);

        val2 *= val2;
        val2 = warp_reduce_sum(val2);
        if (threadIdx.x % 32==0)
            atomicAdd(&loginfo[1], val2);
    }
    if (tid==31 && blockIdx.y==0) {
        *gamma1 += - *gamma2 + T(1);
        *gamma2 = T(0);
    }
}

template<typename T>
__global__ void update_auxs_odd(T *a1,
        T *b1,
        T *a2,
        T *b2,
        T *gamma1,
        T *gamma2,
        T *loginfo,
        T *phi1,
        T *phi2,
        const T* __restrict__ p,
        const T* __restrict__ q,
        const int nrows,
        const int ncols) {
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    T val1=*gamma1;
    T val2=*gamma2;
    T dval;
    T gamma;

    if (tid < nrows && blockIdx.y==0) {
        gamma = (2*(val2 - T(1)) - val1) / (T) (nrows + ncols);
        val1 = a1[tid];
        dval = p[tid];
        val2 = a2[tid] - dval;
        a1[tid] = val1 - val2;
        val1 = (val1 - 2*val2 + gamma) / (T) ncols;
        phi1[tid] = val1;

        a2[tid] = T(0); // Reset for the next reduction

        dval *= val1;
        val2 *= val2;
        val2 = warp_reduce_sum(val2);
        dval = warp_reduce_sum(dval);
        if (threadIdx.x % 32==0) {
            atomicAdd(&loginfo[0], val2);
            atomicAdd(&loginfo[3], dval);
        }
    }
    if (tid < ncols && blockIdx.y==1) {
        gamma = (2*(val2 - T(1)) - val1) / (T) (nrows + ncols);
        val1 = b1[tid];
        dval = q[tid];
        val2 = b2[tid] - dval;
        b1[tid] = val1 - val2;
        val1 = (val1 - 2*val2 + gamma) / (T) nrows;
        phi2[tid] = val1;

        b2[tid] = T(0);

        dval *= val1;
        val2 *= val2;
        val2 = warp_reduce_sum(val2);
        dval = warp_reduce_sum(dval);
        if (threadIdx.x % 32==0) {
            atomicAdd(&loginfo[1], val2);
            atomicAdd(&loginfo[4], dval);
        }
    }
    if (tid==31 && blockIdx.y==0) {
        *gamma1 += - *gamma2 + T(1);
        *gamma2 = T(0);
    }
}

template<typename T>
__global__ void zeros(T *X, const size_t n) {
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    while (tid < n) {
        X[tid] = T(0);
        tid += gridDim.x * blockDim.x;
    }
}

template<typename T>
__global__ void init_auxs(T *a1,
        T *b1,
        T *a2,
        T *b2,
        T *phi1,
        T *phi2,
        T *common,
        const int nrows,
        const int ncols) {
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < nrows && blockIdx.y==0) {
        a1[tid] = T(0);
        a2[tid] = T(0);
        phi1[tid] = T(0);
    }
    if (tid < ncols && blockIdx.y==1) {
        b1[tid] = T(0);
        b2[tid] = T(0);
        phi2[tid] = T(0);
        if (tid < 16)
            common[tid] = T(0);
    }
}

template<typename T>
__global__ void init_x(T *X,
        const T* __restrict__ C,
        const T* __restrict__ p,
        const T* __restrict__ q,
        const T stepsize,
        const int nrows,
        const int ncols) {
    const int tidy = threadIdx.y + blockIdx.y * blockDim.y;
    int offset;
    __shared__ T q_shared[BLOCK_SIZE];
    const T pval=p[tidy];

    for (int idx=0; idx<WORK_SIZE; idx++) {
        offset = (blockIdx.x * WORK_SIZE + idx) * BLOCK_SIZE;
        if (threadIdx.y + offset < ncols)
            q_shared[threadIdx.y] = q[threadIdx.y + offset];
        __syncthreads();

        if (tidy < nrows) {
            if (BLOCK_SIZE + offset <= ncols) {
                #pragma unroll
                for (int e=0; e<BLOCK_SIZE; e++)
                    X[tidy + (e + offset) * nrows] = pval * q_shared[e]
                        - stepsize * C[tidy + (e + offset) * nrows];
            } else { // Edge blocks
                #pragma unroll
                for (int e=ncols-1; e>=offset; e--)
                    X[tidy + e * nrows] = pval * q_shared[e - offset]
                        - stepsize * C[tidy + e * nrows];
            }
        }
    }
}


template<typename T> void step(T *X,
        T *a1,
        T *b1,
        T *a2,
        T *b2,
        T *gamma1,
        T *gamma2,
        T *phi1,
        T *phi2,
        T *normsq,
        const T *C,
        const T *p,
        const T *q,
        const T stepsize,
        const int nrows,
        const int ncols,
        const int iteration) {

    dim3 grid_x((ncols + BLOCK_SIZE * WORK_SIZE - 1) / (BLOCK_SIZE * WORK_SIZE),
            (nrows + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 block_x(1, BLOCK_SIZE);
    dim3 grid_auxs((max(nrows, ncols) + BLOCK_SIZE - 1) / BLOCK_SIZE, 2);
    dim3 block_auxs(BLOCK_SIZE);

    if (iteration % 2 == 0) {
        update_x_even_<float> <<<grid_x, block_x>>>(X, a2, b2, gamma2, normsq,
                phi1, phi2, C, stepsize, nrows, ncols);
        update_auxs_even<float> <<<grid_auxs, block_auxs>>>(a1, b1, a2, b2, gamma1, gamma2,
            normsq, phi1, phi2, p, q,  nrows, ncols);
    } else {
        update_x_odd_<float> <<<grid_x, block_x>>>(X, a2, b2, gamma2, normsq,
                phi1, phi2, C, stepsize, nrows, ncols);
        update_auxs_odd<float> <<<grid_auxs, block_auxs>>>(a1, b1, a2, b2, gamma1, gamma2,
            normsq, phi1, phi2, p, q,  nrows, ncols);
    }
}

template<typename T> std::tuple<T, std::vector<T>, std::vector<T>> drot(py::array_t<T> C_input,
        py::array_t<T> p_input,
        py::array_t<T> q_input,
        const int nrows,
        const int ncols,
        const T stepsize,
        const int maxiters,
        const T eps,
        const bool verbose=true,
        const bool log=false) {

    py::buffer_info C_info = C_input.request(), 
    py::buffer_info p_info = p_input.request();
    py::buffer_info q_info = q_input.request();

    if (C_info.ndim != 2)
        throw std::runtime_error("Number of dimensions for C must be two");

    if (p_info.ndim != 1 || q_info.ndim != 1)
        throw std::runtime_error("Number of dimensions for p and q must be one");

    const T *C = static_cast<T *>(C_info.ptr);
    const T *p = static_cast<T *>(p_info.ptr);
    const T *q = static_cast<T *>(q_info.ptr);

    const size_t matsize = nrows*ncols*sizeof(T);
    const size_t rowsize = nrows*sizeof(T);
    const size_t colsize = ncols*sizeof(T);

    // Initilizing
    T *dC;
    T *dp;
    T *dq;
    T *dX;
    T *da1;
    T *da2;
    T *db1;
    T *db2;
    T *dphi1;
    T *dphi2;
    T *dcommon; // Loginfo and auxiuliary scalars

    hipMalloc((void**)&dC, matsize);
    hipMalloc((void**)&dp, rowsize);
    hipMalloc((void**)&dq, colsize);
    hipMemcpy(dC, &C[0], matsize, hipMemcpyHostToDevice);
    hipMemcpy(dp, &p[0], rowsize, hipMemcpyHostToDevice);
    hipMemcpy(dq, &q[0], colsize, hipMemcpyHostToDevice);

    hipMalloc((void**)&dX, matsize);
    hipMalloc((void**)&da1, rowsize);
    hipMalloc((void**)&da2, rowsize);
    hipMalloc((void**)&db1, colsize);
    hipMalloc((void**)&db2, colsize);
    hipMalloc((void**)&dphi1, rowsize);
    hipMalloc((void**)&dphi2, colsize);
    hipMalloc((void**)&dcommon, 16*sizeof(T));

    dim3 grid_auxs((max(nrows, ncols) + BLOCK_SIZE - 1) / BLOCK_SIZE, 2);
    dim3 block_auxs(BLOCK_SIZE);

    zeros<T> <<<1024, 256>>>(dX, nrows*ncols); // Much faster than creating
    init_auxs<T> <<<grid_auxs, block_auxs>>>(da1, db1, da2, db2, // and
            dphi1, dphi2, dcommon, nrows, ncols); // copying from CPU

    dim3 grid_x((ncols + BLOCK_SIZE * WORK_SIZE - 1) / (BLOCK_SIZE * WORK_SIZE),
            (nrows + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 block_x(1, BLOCK_SIZE);

    init_x<T> <<<block_x, grid_x>>>(dX, dC, dp, dq, stepsize, nrows, ncols);
    hipDeviceSynchronize();

    // Optimizing
    int k = 0;
    bool done = false;
    T loginfo[5];
    T res = 0;
    T fval = 0;

    std::vector<T> residuals;
    std::vector<T> objectives;
    if (log) {
        residuals.reserve(maxiters + 1);
        objectives.reserve(maxiters + 1);
    }
    if (verbose)
        printf("%8s %10s %10s\n", "Iter", "Time", "Residual");
    while ((!done) && (k < maxiters)) {
        step(dX, da1, db1, da2, db2, &dcommon[14], &dcommon[15], dphi1, dphi2,
               &dcommon[0], dC, dp, dq, stepsize, nrows, ncols, k);

        hipMemcpy(&loginfo[0], &dcommon[0], 5*sizeof(T), hipMemcpyDeviceToHost);
        res = sqrt(loginfo[0] + loginfo[1]);
        fval = k % 2 == 0 ? fval: loginfo[2];
        //fval = k % 2 == 0 ? fval: abs(loginfo[2] - (loginfo[3] + loginfo[4]) / stepsize);
        done = res <= eps ? true : false;
        k++;

        if (log) {
            residuals.push_back(res);
            objectives.push_back(fval);
        }
        if (verbose)
            printf("%8d %10.5f %10.5f \n", k, res);
    }

    // Cleaning
    hipFree(dC);
    hipFree(dp);
    hipFree(dq);
    hipFree(dX);
    hipFree(da1);
    hipFree(da2);
    hipFree(db1);
    hipFree(db2);
    hipFree(dphi1);
    hipFree(dphi2);
    hipFree(dcommon);

    return std::make_tuple(fval, residuals, objectives);
}

PYBIND11_MODULE(fast_ot, m)
{
  m.def("drot", &drot<float>, "Douglas-Rachford Splitting for Optimal Transport");
}